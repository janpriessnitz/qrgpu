
#include "QRReference.h"

#include <cstdio>
#include <chrono>

#include "Matrix.h"

// Taken from https://docs.nvidia.com/cuda/cusolver/index.html#ormqr-example1
Matrix QRReferenceCuSolver(const Matrix &A, uint64_t *us_taken) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;

    // cuda algorithms use matrix representation with row as leading dimension
    auto AT = A.getT();
    const int cols = A.cols;
    const int rows = A.rows;
    const int lda = rows;
    /*      | 1 2 3 |
    *   A = | 4 5 6 |
    *       | 2 1 1 |
    *       | 1 1 10 |
    */

/* device memory */
    real *d_A = NULL;
    real *d_tau = NULL;
    int *devInfo = NULL;
    real *d_work = NULL;
    int  lwork_geqrf = 0;
    int  lwork = 0;

/* step 1: create cudense/cublas handle */
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

/* step 2: copy A and B to device */
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(real) * lda * cols);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(real) * cols);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, AT.data, sizeof(real) * lda * cols, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    /* step 3: query working space of geqrf and ormqr */
    cusolver_status = cusolverDn_geqrf_bufferSize(
        cusolverH,
        rows,
        cols,
        d_A,
        lda,
        &lwork_geqrf);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    lwork = lwork_geqrf;

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(real)*lwork);
    assert(hipSuccess == cudaStat1);

    hipDeviceSynchronize();
    auto cuStart = std::chrono::high_resolution_clock::now();


/* step 4: compute QR factorization */
    cusolver_status = cusolverDn_geqrf(
        cusolverH,
        rows,
        cols,
        d_A,
        lda,
        d_tau,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    hipDeviceSynchronize();
    auto cuEnd = std::chrono::high_resolution_clock::now();
    auto cuDuration = std::chrono::duration_cast<std::chrono::microseconds>(cuEnd - cuStart).count();
    *us_taken = cuDuration;

    cudaStat1 = hipMemcpy(AT.data, d_A, sizeof(real) * lda * cols, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    // real taus[100];
    // cudaStat1 = hipMemcpy(taus, d_tau, sizeof(real) * cols, hipMemcpyDeviceToHost);
    // assert(hipSuccess == cudaStat1);

    // printf("cuSolver:\n");
    // AT.getT().print();
    // printf("taus:\n");
    // for (int i = 0; i < cols; ++i) {
    //     printf("%.4f ", taus[i]);
    // }
    // printf("\n");

    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);
    hipDeviceReset();

    return AT.getT();
}
