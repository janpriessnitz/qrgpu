#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <utility>

#include "Matrix.h"
#include "QRKernel.h"
#include "BlockHouseholderKernel.h"

#include "QRInvoke.h"

void InvokeSolve(Matrix *A, real *taus, int cols, uint64_t *usec_taken) {
  int rows = A->rows;
  int cols_expanded = A->cols;
  auto AT = A->getT();
  real *dA = NULL;
  real *dTaus = NULL;

  int device = 0;
  if (hipSetDevice(device) != hipSuccess){
      fprintf(stderr, "Cannot set CUDA device!\n");
      exit(1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  printf("Using device %d: \"%s\"\n", device, deviceProp.name);

  // allocate and set device memory
  if (hipMalloc((void**)&dA, rows*cols_expanded*sizeof(dA[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  hipMemcpy(dA, AT.data, rows*cols_expanded*sizeof(dA[0]), hipMemcpyHostToDevice);

  if (hipMalloc((void**)&dTaus, sizeof(real)*cols) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      return;
  }

  hipDeviceSynchronize();
  QRBlockSolve(dA, dTaus, rows, cols, rows, usec_taken);
  // QRSolve(dA, rows, cols, cols_expanded - cols, rows);
  printf("Kernel launch error: %s\n", hipGetErrorString(hipGetLastError()));
  hipDeviceSynchronize();

  printf("Copying back to resulting matrices\n");

  // hipMemcpy(taus, dTaus, cols*sizeof(real), hipMemcpyDeviceToHost);
  hipMemcpy(AT.data, dA, rows*cols_expanded*sizeof(dA[0]), hipMemcpyDeviceToHost);
  *A = AT.getT();
  cleanup:
    if (dA) hipFree(dA);
    if (dTaus) hipFree(dTaus);
    hipDeviceReset();
}
