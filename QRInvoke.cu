#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <utility>

#include "Matrix.h"
#include "QRKernel.h"

#include "QRInvoke.h"

std::pair<Matrix, Matrix> Invoke(const Matrix &A) {
  int n = A.rows;
  int m = A.cols;
  Matrix Q(n, n);
  Matrix R(n, m);

  double *dA = NULL;
  double *dQ = NULL;
  double *dR = NULL;

  int device = 0;
  if (hipSetDevice(device) != hipSuccess){
      fprintf(stderr, "Cannot set CUDA device!\n");
      exit(1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  printf("Using device %d: \"%s\"\n", device, deviceProp.name);

  // allocate and set device memory
  if (hipMalloc((void**)&dA, n*m*sizeof(dA[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  if (hipMalloc((void**)&dQ, n*n*sizeof(dQ[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  if (hipMalloc((void**)&dR, n*m*sizeof(dR[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  hipMemcpy(dA, A.data, n*m*sizeof(dA[0]), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  QRHouseholder(dA, n, m, dQ, dR);
  printf("Kernel launch error: %s\n", hipGetErrorString(hipGetLastError()));
  hipDeviceSynchronize();

  printf("Copying back to resulting matrices\n");

  hipMemcpy(Q.data, dQ, n*n*sizeof(dQ[0]), hipMemcpyDeviceToHost);
  hipMemcpy(R.data, dR, n*m*sizeof(dR[0]), hipMemcpyDeviceToHost);
  cleanup:
    if (dA) hipFree(dA);
    if (dQ) hipFree(dQ);
    if (dR) hipFree(dR);

  return std::pair<Matrix, Matrix>(Q, R);
}

void InvokeSolve(Matrix *A, int cols) {
  int n = A->rows;
  int m = A->cols;

  double *dA = NULL;

  int device = 0;
  if (hipSetDevice(device) != hipSuccess){
      fprintf(stderr, "Cannot set CUDA device!\n");
      exit(1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  printf("Using device %d: \"%s\"\n", device, deviceProp.name);

  // allocate and set device memory
  if (hipMalloc((void**)&dA, n*m*sizeof(dA[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  hipMemcpy(dA, A->data, n*m*sizeof(dA[0]), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  QRSolve(dA, n, cols, m);
  printf("Kernel launch error: %s\n", hipGetErrorString(hipGetLastError()));
  hipDeviceSynchronize();

  printf("Copying back to resulting matrices\n");

  hipMemcpy(A->data, dA, n*m*sizeof(dA[0]), hipMemcpyDeviceToHost);
  cleanup:
    if (dA) hipFree(dA);

}
