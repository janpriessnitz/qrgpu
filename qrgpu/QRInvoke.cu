#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <utility>

#include "Matrix.h"
#include "QRKernel.h"
#include "BlockHouseholderKernel.h"

#include "QRInvoke.h"

Matrix InvokeSolve(Matrix *A, real *taus, int cols, uint64_t *usec_taken) {
  int rows = A->rows;
  int cols_expanded = A->cols;
  auto AT = A->getT();
  real *dA = NULL;
  real *dTaus = NULL;
  Matrix QR(A->cols, A->rows);

  int device = 0;
  if (hipSetDevice(device) != hipSuccess){
      fprintf(stderr, "Cannot set CUDA device!\n");
      exit(1);
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  // allocate and set device memory
  if (hipMalloc((void**)&dA, rows*cols_expanded*sizeof(dA[0])) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }
  hipMemcpy(dA, AT.data, rows*cols_expanded*sizeof(dA[0]), hipMemcpyHostToDevice);

  if (hipMalloc((void**)&dTaus, sizeof(real)*cols) != hipSuccess) {
      fprintf(stderr, "Device memory allocation error!\n");
      goto cleanup;
  }

  hipDeviceSynchronize();
  QRBlockSolve(dA, dTaus, rows, cols, rows, usec_taken);
  // QRSolve(dA, rows, cols, cols_expanded - cols, rows);
  hipDeviceSynchronize();

  // hipMemcpy(taus, dTaus, cols*sizeof(real), hipMemcpyDeviceToHost);
  hipMemcpy(QR.data, dA, rows*cols_expanded*sizeof(dA[0]), hipMemcpyDeviceToHost);
  cleanup:
    if (dA) hipFree(dA);
    if (dTaus) hipFree(dTaus);
    hipDeviceReset();

  return QR.getT();
}
